#include "hip/hip_runtime.h"
// Copyright 2004-present Facebook. All Rights Reserved.
// Author: Benjamin Graham <benjamingraham@fb.com>

// Tensor formats
// Input: ilen * batchSize * inputPlanes
// Output: olen * batchSize * outputPlanes
// Weight: kw * inputPlanes * outputPlanes

#include <THCGeneral.h>
#include "THCDeviceTensor.cuh"
#include "THCDeviceTensorUtils.cuh"
#include "THCDeviceUtils.cuh"
#include "THCBlas.h"
#include "THCNumerics.cuh"
#include "THCAtomics.cuh"
#include <algorithm>
#include <hip/hip_runtime.h>

namespace detail {

template <typename T>
void Xgemm(THCState *state,
                      char transa,
                      char transb,
                      long m,
                      long n,
                      long k,
                      T alpha,
                      T *a,
                      long lda,
                      T *b,
                      long ldb,
                      T beta,
                      T *c,
                      long ldc);

template<>
void Xgemm<float>(THCState *state,
                  char transa,
                  char transb,
                  long m,
                  long n,
                  long k,
                  float alpha,
                  float *a,
                  long lda,
                  float *b,
                  long ldb,
                  float beta,
                  float *c,
                  long ldc) {
  THCudaBlas_Sgemm(state, transa, transb, m, n, k, alpha,
                   a, lda, b, ldb, beta, c, ldc);
}

template<>
void Xgemm<double>(THCState *state,
                  char transa,
                  char transb,
                  long m,
                  long n,
                  long k,
                  double alpha,
                  double *a,
                  long lda,
                  double *b,
                  long ldb,
                  double beta,
                  double *c,
                  long ldc) {
  THCudaBlas_Dgemm(state, transa, transb, m, n, k, alpha,
                   a, lda, b, ldb, beta, c, ldc);
}

// kernels for forwarding and backwarding bias
template <typename T>
__global__ void TemporalConvolutionTBC_fp_bias(
    T* output_features,
    T* bias,
    int output_stride,
    int rows) {
  int x = blockIdx.x * 32 + threadIdx.x;
  T b = bias[x];
  for (int row = blockIdx.y; row < rows; row += gridDim.y) {
    output_features[row * output_stride + x] = b;
  }
}

template <typename T, typename AccT>
__global__ void TemporalConvolutionTBC_bp_bias(
    T* matrix,
    T* target,
    int rows,
    int stride,
    AccT scale) {
  int i = blockIdx.x * 32 + threadIdx.x;
  AccT t = 0;
  for (int j = blockIdx.y; j < rows; j += gridDim.y)
    t += matrix[j * stride + i];
  atomicAdd(&target[i], t * scale);
}

template <typename T>
void runTemporalConvolutionTBC_updateOutput(
    THCState* state,
    const THCDeviceTensor<T, 3>& input,
    const THCDeviceTensor<T, 3>& output,
    const THCDeviceTensor<T, 3>& weight,
    const THCDeviceTensor<T, 1>& bias) {

  auto W = weight.data();
  auto B = bias.data();
  auto I = input.data();
  auto O = output.data();

  auto ilen = input.getSize(0);
  auto batchSize = input.getSize(1);
  auto inputPlanes = input.getSize(2);
  auto outputPlanes = output.getSize(2);
  auto olen = output.getSize(0);
  auto kw = weight.getSize(0);
  int pad = (olen - ilen + kw - 1) / 2;

  // input * weights + bias -> output_features

  int op32n = outputPlanes / 32;
  int op32r = outputPlanes % 32;
  if (op32n) {
    TemporalConvolutionTBC_fp_bias<<<
        dim3(op32n, 32),
        32,
        0,
        THCState_getCurrentStream(state)>>>(
          O, B, output.getStride(1), batchSize * olen);
  }
  if (op32r) {
    TemporalConvolutionTBC_fp_bias<<<
        dim3(1, 32),
        op32r,
        0,
        THCState_getCurrentStream(state)>>>(
          O + op32n * 32, B + op32n * 32, output.getStride(1), batchSize * olen);
  }

  for (int k = 0; k < kw; k++) {
    int iShift = std::max(0, k - pad);
    int oShift = std::max(0, pad - k);
    int t = std::min(ilen + pad - k, olen) - oShift;
    // Note: gemm assumes column-major matrices
    // input    is l*m (row-major)
    // weight   is m*r (row-major)
    // output   is l*r (row-major)
    if (t > 0)
       Xgemm<T>(
          state,
          'n',
          'n',
          outputPlanes, // r
          batchSize * t, // l
          inputPlanes, // m
          1, // alpha
          W + k * weight.getStride(0),
          outputPlanes, // r
          I + iShift * input.getStride(0),
          input.getStride(1), // >=m
          1, // beta
          O + oShift * output.getStride(0),
          output.getStride(1) // r
        );
  }
}

template <typename T>
void runTemporalConvolutionTBC_updateGradInput(
    THCState* state,
    const THCDeviceTensor<T, 3>& dInput,
    const THCDeviceTensor<T, 3>& dOutput,
    const THCDeviceTensor<T, 3>& weight) {
  auto ilen = dInput.getSize(0);
  auto batchSize = dInput.getSize(1);
  auto inputPlanes = dInput.getSize(2);
  auto outputPlanes = dOutput.getSize(2);
  auto olen = dOutput.getSize(0);
  auto kw = weight.getSize(0);
  int pad = (olen - ilen + kw - 1) / 2;

  auto W = weight.data();
  auto dI = dInput.data();
  auto dO = dOutput.data();

  for (int k = 0; k < kw; k++) {
    int iShift = std::max(0, k - pad);
    int oShift = std::max(0, pad - k);
    int t = std::min(ilen + pad - k, olen) - oShift;
    // dOutput * T(weight) -> dInput
    // Note: gemm assumes column-major matrices
    // dOutput is l*m (row-major)
    // weight  is r*m (row-major)
    // dInput  is l*r (row-major)
    if (t > 0)
      Xgemm<T>(
          state,
          't',
          'n',
          inputPlanes, // r
          batchSize * t, // l
          outputPlanes, // m
          1, // alpha
          W + k * weight.getStride(0),
          outputPlanes, // m
          dO + oShift * dOutput.getStride(0),
          dOutput.getStride(1), // m
          1, // beta
          dI + iShift * dInput.getStride(0),
          dInput.getStride(1) // m
        );
  }
}

template <typename T, typename AccT>
void runTemporalConvolutionTBC_accGradParameters(
    THCState* state,
    const THCDeviceTensor<T, 3>& input,
    const THCDeviceTensor<T, 3>& dOutput,
    const THCDeviceTensor<T, 3>& dWeight,
    const THCDeviceTensor<T, 1>& dBias,
    AccT scale) {
  auto ilen = input.getSize(0);
  auto batchSize = input.getSize(1);
  auto inputPlanes = input.getSize(2);
  auto outputPlanes = dOutput.getSize(2);
  auto olen = dOutput.getSize(0);
  auto kw = dWeight.getSize(0);
  int pad = (olen - ilen + kw - 1) / 2;

  auto dW = dWeight.data();
  auto dB = dBias.data();
  auto I = input.data();
  auto dO = dOutput.data();

  int op32n = outputPlanes / 32;
  int op32r = outputPlanes % 32;
  if (op32n) {
    TemporalConvolutionTBC_bp_bias<<<
        dim3(op32n, 32),
        32,
        0,
        THCState_getCurrentStream(state)>>>(
          dO, dB, batchSize * olen, dOutput.getStride(1), scale);
  }
  if (op32r) {
    TemporalConvolutionTBC_bp_bias<<<
        dim3(1, 32),
        op32r,
        0,
        THCState_getCurrentStream(state)>>>(
        dO + op32n * 32,
        dB + op32n * 32,
        batchSize * olen,
        dOutput.getStride(1),
        scale);
  }

  for (int k = 0; k < kw; k++) {
    int iShift = std::max(0, k - pad);
    int oShift = std::max(0, pad - k);
    int t = std::min(ilen + pad - k, olen) - oShift;
    // Note: gemm assumes column-major matrices
    // Input    is m*l (row-major)
    // dOutput  is m*r (row-major)
    // dWeight  is l*r (row-major)
    if (t > 0)
      Xgemm<T>(
          state,
          'n',
          't',
          outputPlanes, // r
          inputPlanes, // l
          batchSize * t, // m
          scale, // alpha
          dO + oShift * dOutput.getStride(0),
          dOutput.getStride(1), // r
          I + iShift * input.getStride(0),
          input.getStride(1), // l
          1, // beta
          dW + k * dWeight.getStride(0),
          outputPlanes // r
        );
   }
}
} // namespaces

#include "generic/TemporalConvolutionTBCHost.cu"
#include "THCGenerateFloatType.h"

#include "generic/TemporalConvolutionTBCHost.cu"
#include "THCGenerateDoubleType.h"
