#include "hip/hip_runtime.h"
// Copyright 2004-present Facebook. All Rights Reserved.
// Author: Benjamin Graham <benjamingraham@fb.com>

// Tensor formats
// Input: ilen * batchSize * inputPlanes
// Output: olen * batchSize * outputPlanes
// Weight: kw * inputPlanes * outputPlanes

#include "TemporalConvolutionTBC.cuh"
#include "THCBlas.h"

namespace detail {

// kernels for forwarding and backwarding bias
__global__ void TemporalConvolutionTBC_fp_bias(
    float* output_features,
    float* bias,
    int output_stride,
    int rows) {
  int x = blockIdx.x * 32 + threadIdx.x;
  float b = bias[x];
  for (int row = blockIdx.y; row < rows; row += gridDim.y) {
    output_features[row * output_stride + x] = b;
  }
}
__global__ void TemporalConvolutionTBC_bp_bias(
    float* matrix,
    float* target,
    int rows,
    int stride,
    float scale) {
  int i = blockIdx.x * 32 + threadIdx.x;
  float t = 0;
  for (int j = blockIdx.y; j < rows; j += gridDim.y)
    t += matrix[j * stride + i];
  atomicAdd(&target[i], t * scale);
}

void runTemporalConvolutionTBC_updateOutput(
    THCState* state,
    const THCDeviceTensor<float, 3>& input,
    const THCDeviceTensor<float, 3>& output,
    const THCDeviceTensor<float, 3>& weight,
    const THCDeviceTensor<float, 1>& bias) {

  auto W = weight.data();
  auto B = bias.data();
  auto I = input.data();
  auto O = output.data();

  auto ilen = input.getSize(0);
  auto batchSize = input.getSize(1);
  auto inputPlanes = input.getSize(2);
  auto outputPlanes = output.getSize(2);
  auto olen = output.getSize(0);
  auto kw = weight.getSize(0);
  int pad = (olen - ilen + kw - 1) / 2;

  // input * weights + bias -> output_features

  int op32n = outputPlanes / 32;
  int op32r = outputPlanes % 32;
  if (op32n) {
    TemporalConvolutionTBC_fp_bias<<<
        dim3(op32n, 32),
        32,
        0,
        THCState_getCurrentStream(state)>>>(
          O, B, output.getStride(1), batchSize * olen);
  }
  if (op32r) {
    TemporalConvolutionTBC_fp_bias<<<
        dim3(1, 32),
        op32r,
        0,
        THCState_getCurrentStream(state)>>>(
          O + op32n * 32, B + op32n * 32, output.getStride(1), batchSize * olen);
  }

  for (int k = 0; k < kw; k++) {
    int iShift = std::max(0, k - pad);
    int oShift = std::max(0, pad - k);
    int t = std::min(ilen + pad - k, olen) - oShift;
    // Note: gemm assumes column-major matrices
    // input    is l*m (row-major)
    // weight   is m*r (row-major)
    // output   is l*r (row-major)
    if (t > 0)
       THCudaBlas_Sgemm(
          state,
          'n',
          'n',
          outputPlanes, // r
          batchSize * t, // l
          inputPlanes, // m
          1, // alpha
          W + k * weight.getStride(0),
          outputPlanes, // r
          I + iShift * input.getStride(0),
          input.getStride(1), // >=m
          1, // beta
          O + oShift * output.getStride(0),
          output.getStride(1) // r
        );
  }
}

void runTemporalConvolutionTBC_updateGradInput(
    THCState* state,
    const THCDeviceTensor<float, 3>& dInput,
    const THCDeviceTensor<float, 3>& dOutput,
    const THCDeviceTensor<float, 3>& weight) {
  auto ilen = dInput.getSize(0);
  auto batchSize = dInput.getSize(1);
  auto inputPlanes = dInput.getSize(2);
  auto outputPlanes = dOutput.getSize(2);
  auto olen = dOutput.getSize(0);
  auto kw = weight.getSize(0);
  int pad = (olen - ilen + kw - 1) / 2;

  auto W = weight.data();
  auto dI = dInput.data();
  auto dO = dOutput.data();

  for (int k = 0; k < kw; k++) {
    int iShift = std::max(0, k - pad);
    int oShift = std::max(0, pad - k);
    int t = std::min(ilen + pad - k, olen) - oShift;
    // dOutput * T(weight) -> dInput
    // Note: gemm assumes column-major matrices
    // dOutput is l*m (row-major)
    // weight  is r*m (row-major)
    // dInput  is l*r (row-major)
    if (t > 0)
      THCudaBlas_Sgemm(
          state,
          't',
          'n',
          inputPlanes, // r
          batchSize * t, // l
          outputPlanes, // m
          1, // alpha
          W + k * weight.getStride(0),
          outputPlanes, // m
          dO + oShift * dOutput.getStride(0),
          dOutput.getStride(1), // m
          1, // beta
          dI + iShift * dInput.getStride(0),
          dInput.getStride(1) // m
          );
  }
}

void runTemporalConvolutionTBC_accGradParameters(
    THCState* state,
    const THCDeviceTensor<float, 3>& input,
    const THCDeviceTensor<float, 3>& dOutput,
    const THCDeviceTensor<float, 3>& dWeight,
    const THCDeviceTensor<float, 1>& dBias,
    float scale) {
  auto ilen = input.getSize(0);
  auto batchSize = input.getSize(1);
  auto inputPlanes = input.getSize(2);
  auto outputPlanes = dOutput.getSize(2);
  auto olen = dOutput.getSize(0);
  auto kw = dWeight.getSize(0);
  int pad = (olen - ilen + kw - 1) / 2;

  auto dW = dWeight.data();
  auto dB = dBias.data();
  auto I = input.data();
  auto dO = dOutput.data();

  int op32n = outputPlanes / 32;
  int op32r = outputPlanes % 32;
  if (op32n) {
    TemporalConvolutionTBC_bp_bias<<<
        dim3(op32n, 32),
        32,
        0,
        THCState_getCurrentStream(state)>>>(
          dO, dB, batchSize * olen, dOutput.getStride(1), scale);
  }
  if (op32r) {
    TemporalConvolutionTBC_bp_bias<<<
        dim3(1, 32),
        op32r,
        0,
        THCState_getCurrentStream(state)>>>(
        dO + op32n * 32,
        dB + op32n * 32,
        batchSize * olen,
        dOutput.getStride(1),
        scale);
  }

  for (int k = 0; k < kw; k++) {
    int iShift = std::max(0, k - pad);
    int oShift = std::max(0, pad - k);
    int t = std::min(ilen + pad - k, olen) - oShift;
    // Note: gemm assumes column-major matrices
    // Input    is m*l (row-major)
    // dOutput  is m*r (row-major)
    // dWeight  is l*r (row-major)
    if (t > 0)
      THCudaBlas_Sgemm(
          state,
          'n',
          't',
          outputPlanes, // r
          inputPlanes, // l
          batchSize * t, // m
          scale, // alpha
          dO + oShift * dOutput.getStride(0),
          dOutput.getStride(1), // r
          I + iShift * input.getStride(0),
          input.getStride(1), // l
          1, // beta
          dW + k * dWeight.getStride(0),
          outputPlanes // r
          );
   }
}
} // namespaces

#include "generic/TemporalConvolutionTBCHost.cu"
#include "THCGenerateFloatType.h"
