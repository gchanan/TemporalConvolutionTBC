#include <THCGeneral.h>

#include <algorithm>
#include <hip/hip_runtime.h>
#include "THCDeviceTensor.cuh"
#include "THCDeviceTensorUtils.cuh"
#include "THCDeviceUtils.cuh"
#include "TemporalConvolutionTBC.cuh"

#define TemporalConvolutionTBC_CONCAT_EXPAND(x,y,z) x ## y ## _ ## z
#define TemporalConvolutionTBC_CONCAT(x,y,z) TemporalConvolutionTBC_CONCAT_EXPAND(x,y,z)
#define TemporalConvolutionTBC_(NAME) TemporalConvolutionTBC_CONCAT(TemporalConvolutionTBC_, NAME, CReal)

#include "TemporalConvolutionTBC.cu"
//#include "THCGenerateFloatType.h"

//#include "TemporalConvolutionTBCHost.cpp"
//#include "THCGenerateDoubleType.h"
